#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2019 Tokyo Institute of Technology
 */

//============================================================================//
//
//  Software Name : MEGADOCK
//
//  Class Name : FFTProcessTable
//
//  Contact address : Tokyo Institute of Technology, AKIYAMA Lab.
//
//============================================================================//

#include "fft_process_table.h"

#define NUM_THREADS 512 //should be power of 2

#ifdef CUFFT

#include "cuda_kernel.h"

#endif

//============================================================================//
void FFTProcessTable::alloc_array(const int &num_fft)
//============================================================================//
{
    //cout << "FFT::alloc_array |" <<num_fft<< endl; cout.flush();
    _Num_fft = num_fft;

    const size_t nf3       = _Num_fft * _Num_fft * _Num_fft;
    const int num_sort  = _parameter->_Num_sort;
    const int num_angle = _parameter->_Num_rot_angles;
    const int no        = _parameter->_Num_output;
    const size_t nproc2    = _parallel->nproc2();
    int   num_toprank;

    num_toprank = num_angle * num_sort;
    if( no > num_toprank ) num_toprank = no;

    alloc_fft();

    _Select.resize(num_sort);

    _Top.resize(num_toprank);

    //---------- memory allocation for _Current_rot_angle_num
    //_Current_rot_angle_num = new int[nproc2];

    _exec_logger->record_malloc( sizeof(float)*nf3*2*(1 + nproc2));

    //---------- memory allocation for _FFT_rec_r
    _FFT_rec_r = new float[nf3];
    if( !_FFT_rec_r ) {
        cerr << "[ERROR] Out of memory. Number of listed receptors = ("
             << nf3 << ") for (_FFT_rec_r) in fft_process.cpp!!\n";
        exit(1);
    }

    //---------- memory allocation for _FFT_rec_i
    _FFT_rec_i = new float[nf3];
    if( !_FFT_rec_i ) {
        cerr << "[ERROR] Out of memory. Number of listed receptors = ("
             << nf3 << ") for (_FFT_rec_i) in fft_process.cpp!!\n";
        exit(1);
    }

    return;
}

//============================================================================//
void FFTProcessTable::alloc_fft()
//============================================================================//
{
    const int nf1 = _Num_fft;
    const size_t nf3 = _Num_fft * _Num_fft * _Num_fft;
    const size_t nproc2  = _parallel->nproc2();
    const int num_gpu = _parallel->num_gpu();
    const int na = _ligand->num_atoms();

#ifdef CUFFT
    const int num_sort = _parameter->_Num_sort;
    const int ng1 = _Num_fft / 2;
    const int ng3 = ng1 * ng1 * ng1;
    const int nag = na * ng1;
    //for ligand voxelization on GPU
    const int nThreads = NUM_THREADS;
    const int nBlocks_nf3 = (nf3 + (nThreads-1)) / nThreads;

    CUFFTin_host  = new hipfftComplex[nf3];
    CUFFTout_host = new hipfftComplex[nf3];

    _exec_logger->record_malloc( sizeof(hipfftComplex)*nf3*2 ); //_in/outBuf

    //printf(" start: %p\n",&CUFFTin_host[0].x);

    int lenCUFFTin_host = (int)(((long int)&CUFFTin_host[nf3-1].x) - ((long int)&CUFFTin_host[0].x) + sizeof(CUFFTin_host[nf3-1]))/sizeof(CUFFTin_host[nf3-1]);
    if(lenCUFFTin_host !=nf3) printf("# discontinuous memory allocation occurs\n");

    //printf("   end: %ld\n",(long long int)&CUFFTin_host[nf3-1].y - &CUFFTin_host[0].x);

    int myid2 = omp_get_thread_num();
    hipSetDevice(myid2 % num_gpu);
    checkCudaErrors( hipStreamCreate(&_cuda_stream));
    cufft_result = hipfftPlan3d(&cufft_plan, nf1, nf1, nf1, HIPFFT_C2C);
    hipfftSetStream(cufft_plan, _cuda_stream);

    checkCudaErrors( hipMalloc((void **)&CUFFTin_gpu,  sizeof(hipfftComplex)*nf3) );
    checkCudaErrors( hipMalloc((void **)&CUFFTout_gpu, sizeof(hipfftComplex)*nf3) );
    checkCudaErrors( hipMalloc((void **)&_FFT_rec_r_gpu, sizeof(float)*nf3) );
    checkCudaErrors( hipMalloc((void **)&_FFT_rec_i_gpu, sizeof(float)*nf3) );

    checkCudaErrors( hipMalloc((void **)&grid_r_gpu,  sizeof(float)*ng3));
    checkCudaErrors( hipMalloc((void **)&grid_i_gpu,  sizeof(float)*ng3));
    checkCudaErrors( hipMalloc((void **)&grid_coord_gpu,  sizeof(float)*ng1));
    checkCudaErrors( hipMalloc((void **)&radius_core2_gpu,  sizeof(float)*na));
    checkCudaErrors( hipMalloc((void **)&radius_surf2_gpu,  sizeof(float)*na));
    checkCudaErrors( hipMalloc((void **)&_Charge_gpu,  sizeof(float)*na));
    checkCudaErrors( hipMalloc((void **)&xd_gpu,  sizeof(float)*nag));
    checkCudaErrors( hipMalloc((void **)&yd_gpu,  sizeof(float)*nag));
    checkCudaErrors( hipMalloc((void **)&zd_gpu,  sizeof(float)*nag));
    checkCudaErrors( hipMalloc((void **)&atom_coord_rotated_gpu,  sizeof(float)*na*3));
    checkCudaErrors( hipMalloc((void **)&atom_coord_orig_gpu,  sizeof(float)*na*3));
    checkCudaErrors( hipMalloc((void **)&mole_center_coord_gpu,  sizeof(float)*3));
    checkCudaErrors( hipMalloc((void **)&ligand_rotation_angle_gpu,  sizeof(float)*3));
    checkCudaErrors( hipMalloc((void **)&top_score_gpu, sizeof(float)*nBlocks_nf3*num_sort) );
    checkCudaErrors( hipMalloc((void **)&top_index_gpu, sizeof(int)*nBlocks_nf3*num_sort) );
    top_score_host = new float[nBlocks_nf3];
    top_index_host = new int[nBlocks_nf3];

    _exec_logger->record_malloc( sizeof(float)*nBlocks_nf3 + sizeof(int)*nBlocks_nf3 );

    hipMemGetInfo(&(_exec_logger->devmem_free), &(_exec_logger->devmem_total));
    _exec_logger->devmem_use = _exec_logger->devmem_total - _exec_logger->devmem_free;

#else

    _FFTWin  = (fftwf_complex*) fftwf_malloc(sizeof(fftwf_complex)*nf3);
    _FFTWout = (fftwf_complex*) fftwf_malloc(sizeof(fftwf_complex)*nf3);

    plan_fftw_forward=fftwf_plan_dft_3d(nf1,nf1,nf1,_FFTWin,_FFTWout,FFTW_FORWARD,FFTW_ESTIMATE);
    plan_fftw_inverse=fftwf_plan_dft_3d(nf1,nf1,nf1,_FFTWin,_FFTWout,FFTW_BACKWARD,FFTW_ESTIMATE);

    _exec_logger->record_malloc( sizeof(fftwf_complex)*nf3*2 );

#endif
    return;
}

//============================================================================//
void FFTProcessTable::receptor_fft(float *grid_r,float *grid_i)
//============================================================================//
{
    const int num_grid= _Num_fft / 2;
    const size_t nf3 = _Num_fft * _Num_fft * _Num_fft;
    const int ndata   = ( _Num_fft - num_grid ) / 2;
    const float   theta   = -2.0 * PI / _Num_fft;

    const int num_gpu = _parallel->num_gpu();
    const int nproc2 = _parallel->nproc2();

    if(num_gpu > 0) {
#ifdef CUFFT
        int myid2 = omp_get_thread_num();
        struct timeval et1, et2;
        //memset(CUFFTin_host[0], make_hipComplex(0.0, 0.0), sizeof(hipfftComplex)*nf3);
        for( int i = 0 ; i < nf3 ; i++ ) {
            CUFFTin_host[i] = make_hipComplex(0.0, 0.0);
        }

        for( int i = 0, m = 0 ; i < num_grid ; i++ ) {
            const int ic = _Num_fft*_Num_fft*(i+ndata);
            for( int j = 0 ; j < num_grid ; j++ ) {
                const int jc = ic + _Num_fft*(j+ndata);
                for( int k = 0 ; k < num_grid ; k++ ) {
                    CUFFTin_host[jc+k+ndata] = make_hipComplex(grid_r[m  ], grid_i[m]);
                    m++;
                }
            }
        }

        hipSetDevice(myid2 % num_gpu); //CUFFTin_dev[0] : [0] means 0th GPU

        gettimeofday(&et1,NULL);
        checkCudaErrors( hipMemcpyAsync(CUFFTin_gpu, CUFFTin_host, sizeof(hipfftComplex)*nf3, hipMemcpyHostToDevice, _cuda_stream) );
        gettimeofday(&et2,NULL);
        _exec_logger->_cputime->t6_data_transfer_rec += (et2.tv_sec-et1.tv_sec + (float)((et2.tv_usec-et1.tv_usec)*1e-6));

        fft3d(theta); // [0] means performed on 0th GPU

        gettimeofday(&et1,NULL);
        checkCudaErrors( hipMemcpyAsync(CUFFTout_host,CUFFTout_gpu,sizeof(hipfftComplex)*nf3,hipMemcpyDeviceToHost, _cuda_stream) );
        gettimeofday(&et2,NULL);
        _exec_logger->_cputime->t6_data_transfer_rec += (et2.tv_sec-et1.tv_sec + (float)((et2.tv_usec-et1.tv_usec)*1e-6));

        for( int i = 0 ; i < nf3 ; i++ ) {
            _FFT_rec_r[i] = hipCrealf(CUFFTout_host[i]);
            _FFT_rec_i[i] = hipCimagf(CUFFTout_host[i]);
        }

        gettimeofday(&et1,NULL);

        checkCudaErrors( hipMemcpyAsync(_FFT_rec_r_gpu, _FFT_rec_r, sizeof(float)*nf3, hipMemcpyHostToDevice, _cuda_stream) );
        checkCudaErrors( hipMemcpyAsync(_FFT_rec_i_gpu, _FFT_rec_i, sizeof(float)*nf3, hipMemcpyHostToDevice, _cuda_stream) );

        gettimeofday(&et2,NULL);
        _exec_logger->_cputime->t6_data_transfer_rec += (et2.tv_sec-et1.tv_sec + (float)((et2.tv_usec-et1.tv_usec)*1e-6));
#endif
    } else {
#ifndef CUFFT
        memset(_FFTWin, 0.0, sizeof(fftwf_complex)*nf3);

        for( int i = 0, m = 0 ; i < num_grid ; i++ ) {
            const int ic = _Num_fft*_Num_fft*(i+ndata);

            for( int j = 0 ; j < num_grid ; j++ ) {
                const int jc = ic + _Num_fft*(j+ndata);

                for( int k = 0 ; k < num_grid ; k++ ) {
                    _FFTWin[jc+k+ndata][0] = grid_r[m  ];
                    _FFTWin[jc+k+ndata][1] = grid_i[m++];
                }
            }
        }

        fft3d(theta);

        for( int i = 0 ; i < nf3 ; i++ ) {
            _FFT_rec_r[i] = _FFTWout[i][0];
            _FFT_rec_i[i] = _FFTWout[i][1];
        }
#endif
    }


    return;
}

#ifndef CUFFT
//============================================================================//
void FFTProcessTable::ligand_preparation(float *grid_r,float *grid_i)
//============================================================================//
{
    const int ng1 = _Num_fft / 2;
    const int nf2 = _Num_fft * _Num_fft;
    const size_t nf3 = _Num_fft * _Num_fft * _Num_fft;
    const int ndata   = ( _Num_fft - ng1 ) / 2;
   
    memset(_FFTWin[0], 0.0, sizeof(fftwf_complex)*nf3);
        
    for( int i = 0, m = 0 ; i < ng1 ; i++ ) {
        const int ic = nf2*(i+ndata);

        for( int j = 0 ; j < ng1 ; j++ ) {
            int jc = ic + _Num_fft*(j+ndata);
            
            for( size_t k = 0, myijk=jc+ndata ; k < ng1 ; k++, myijk++ ) {
                _FFTWin[myijk][0] = grid_r[m  ];
                _FFTWin[myijk][1] = grid_i[m++];
            }
        }
    }
    
    return;
}

//============================================================================//
void FFTProcessTable::convolution()
//============================================================================//
{
    const int nf1 = _Num_fft;
    const int nf2 = nf1*nf1;
    const size_t nf3 = nf1*nf2;

    for( size_t i = 0, j=0 ; i < nf3 ; i++,j++ ) {
      _FFTWin[j][0] = _FFT_rec_r[i]*_FFTWout[j][0] + _FFT_rec_i[i]*_FFTWout[j][1];
      _FFTWin[j][1] = _FFT_rec_r[i]*_FFTWout[j][1] - _FFT_rec_i[i]*_FFTWout[j][0];
    }

    return;
}
#endif

//============================================================================//
void FFTProcessTable::fft3d(const float &theta)
//============================================================================//
{   
    const size_t nproc2  = _parallel->nproc2();
    const int num_gpu = _parallel->num_gpu();

#ifdef CUFFT
    const int nf1 = _Num_fft;
    hipfftHandle plan;
    hipfftResult res;

    res = hipfftPlan3d(&plan, nf1, nf1, nf1, HIPFFT_C2C);
    hipfftSetStream(plan, _cuda_stream);
    if(!res == HIPFFT_SUCCESS) {
        cout << "!fail to plan 3d FFT (DFT):" << res << endl;
        exit(-1);
    }

    if( theta < 0.0 ) {
        res = hipfftExecC2C(plan, CUFFTin_gpu, CUFFTout_gpu, HIPFFT_FORWARD);
    } else {
        res = hipfftExecC2C(plan, CUFFTin_gpu, CUFFTout_gpu, HIPFFT_BACKWARD);
    }

    if(!res == HIPFFT_SUCCESS) {
        cout << "!fail to exec 3d FFT(in fft3d()):" << res << endl;
        exit(-1);
    }

    res =  hipfftDestroy(plan);
#else
    struct timeval et3, et4;
    gettimeofday(&et3,NULL);
    if( _parameter->fft_library_type == 2 ) {        
    } else {
        if( theta < 0.0 ) {
            fftwf_execute(plan_fftw_forward);
        } else {
            fftwf_execute(plan_fftw_inverse);
        }
    }
    gettimeofday(&et4,NULL);
    //printf(" [FFT(host),%s] %10.5f\n\n",((theta<0.0)?"Forward":"Inverse"),(et4.tv_sec-et3.tv_sec + (float)((et4.tv_usec-et3.tv_usec)*1e-6)));
#endif

    return;
}

#ifndef CUFFT
//============================================================================//
void FFTProcessTable::score_sort()
//============================================================================//
{
    const int num_sort  = _parameter->_Num_sort;
    const int nf2 = _Num_fft * _Num_fft;
    const int nf3 = _Num_fft * _Num_fft * _Num_fft;
    float temp_top_score;
    int temp_top_index;

    for( int i = 0 ; i < num_sort ; i++ ) {
        _Select[i].score = -99999.0;
    }

    fftwf_complex *fftout;
    fftout = _FFTWout;
    
    if(num_sort!=1) {
        for( size_t i = 0,myi= 0 ; i < nf3 ; i++,myi++ ) {
            const float raw = fftout[myi][0] / nf3;
            if( raw < _Select[num_sort-1].score) continue;
            for( int j = 0 ; j < num_sort ; j++ ) {
                if( raw > _Select[j].score ) {
                    for( int k = num_sort-1 ; k > j ; k-- ) {
                        _Select[k] = _Select[k-1];
                    }
                    _Select[j].score    = raw;
                    _Select[j].index[1] = i / nf2;
                    _Select[j].index[2] = (i / _Num_fft) % _Num_fft;
                    _Select[j].index[3] = i % _Num_fft;
                    break;
                }
            }
        }
    } else { // num_sort = 1, take only 1 score per angle
        temp_top_score = 0.0;
        temp_top_index = 0;
        for( size_t i = 0, myi=0 ; i < nf3 ; i++,myi++ ) {
            const float raw = fftout[myi][0];
            if (temp_top_score < raw) {
                temp_top_score = raw;
                temp_top_index = i;
            }
        }
        _Select[0].score    = temp_top_score / nf3;
        _Select[0].index[1] = temp_top_index / nf2;
        _Select[0].index[2] = (temp_top_index / _Num_fft) % _Num_fft;
        _Select[0].index[3] = temp_top_index % _Num_fft;
    }

    for( int i = 0 ; i < num_sort ; i++ ) {
        //printf(" top %d %f\n",i,_Select[i].score);
        _Select[i].index[0] = _Current_rot_angle_num;
    }

    for( int i = 0 ; i < num_sort ; i++ ) {
        _Top[_Current_rot_angle_num*num_sort+i] = _Select[i];
    }

    return;
}
#endif

#ifdef CUFFT
//============================================================================//
void FFTProcessTable::cuda_fft(float *grid_r,float *grid_i,float *grid_coord,float *atom_coord_rotated,float *theta, size_t myid2)
//============================================================================//
{
    const int nf1 = _Num_fft;
    const int nf2 = nf1 * nf1;
    const size_t nf3 = nf2 * nf1;
    const int num_gpu = _parallel->num_gpu();
    const size_t nproc2    = _parallel->nproc2();

    const int num_sort = _parameter->_Num_sort;
    const int na = _ligand->num_atoms();

    struct timeval et1, et2;
    struct timeval et3, et4;
    gettimeofday(&et1,NULL);

    float temp_top_score = -999999.0;
    int temp_top_index = -999999;

    const int nThreads = NUM_THREADS;
    const int nBlocks_nf3 = (nf3 + (nThreads-1)) / nThreads;
    if(nBlocks_nf3 * nThreads < nf3) {
        printf(" nf3:%d, nBlocks_nf3:%d, nThreads:%d , nf3=nBlocks_nf3*nThreads\n",nf3,nBlocks_nf3,nThreads);
        fprintf(stderr, " [ERROR] too large FFT size. nf3:%d, nBlocks_nf3:%d\n", nf3, nBlocks_nf3);
        exit(1);
    }

    hipSetDevice(myid2 % num_gpu);
    //printf(" #p10 [myid=%d]\n",myid2);

    ligand_voxelization_on_gpu(theta,myid2);
    checkCudaErrors( hipStreamSynchronize(_cuda_stream) );

    gettimeofday(&et2,NULL);
    _exec_logger->_cputime->t3_1_ligand_voxelization += (et2.tv_sec-et1.tv_sec + (float)((et2.tv_usec-et1.tv_usec)*1e-6));
    gettimeofday(&et1,NULL);

    cufft_result = hipfftExecC2C(cufft_plan, CUFFTin_gpu, CUFFTout_gpu, HIPFFT_FORWARD);
    if(!cufft_result == HIPFFT_SUCCESS) {
        cout << "!fail to exec 3d FFT (DFT, Lig):" << cufft_result << endl;
        exit(-1);
    }

    //*/
    checkCudaErrors( hipStreamSynchronize(_cuda_stream) );

    gettimeofday(&et2,NULL);
    _exec_logger->_cputime->t3_2_fftprocess_ligand_fft += (et2.tv_sec-et1.tv_sec + (float)((et2.tv_usec-et1.tv_usec)*1e-6));

    gettimeofday(&et1,NULL);
    convolution_gpu<<<nBlocks_nf3, nThreads, 0, _cuda_stream>>>(nf3, _FFT_rec_r_gpu, _FFT_rec_i_gpu, CUFFTout_gpu, CUFFTin_gpu);

    checkCudaErrors( hipStreamSynchronize(_cuda_stream) );

    gettimeofday(&et2,NULL);
    _exec_logger->_cputime->t3_3_fftprocess_convolution += (et2.tv_sec-et1.tv_sec + (float)((et2.tv_usec-et1.tv_usec)*1e-6));
    gettimeofday(&et1,NULL);

    cufft_result = hipfftExecC2C(cufft_plan, CUFFTin_gpu, CUFFTout_gpu, HIPFFT_BACKWARD);
    if(!(cufft_result == HIPFFT_SUCCESS)) {
        cout << "!fail to exec 3d FFT (IDFT):" << cufft_result << endl;
        exit(-1);
    }
    //*
    checkCudaErrors( hipStreamSynchronize(_cuda_stream) );
    gettimeofday(&et2,NULL);
    _exec_logger->_cputime->t3_4_fftprocess_fft_inverse += (et2.tv_sec-et1.tv_sec + (float)((et2.tv_usec-et1.tv_usec)*1e-6));
    gettimeofday(&et1,NULL);

    // Search max score translation position from CUFFTout_gpu[nf3]

    //printf(" t=%d per angle\n",num_sort);

    for( int i = 0 ; i < num_sort ; i++ ) {
        _Select[i].score = -99999.0;
    }

    max_pos_single<<<nBlocks_nf3, nThreads, sizeof(float)*nThreads, _cuda_stream>>>(nf3, CUFFTout_gpu,  top_score_gpu, top_index_gpu);
    checkCudaErrors( hipStreamSynchronize(_cuda_stream) );

    gettimeofday(&et3,NULL);
    checkCudaErrors( hipMemcpyAsync(top_score_host,top_score_gpu,sizeof(float)*nBlocks_nf3,hipMemcpyDeviceToHost, _cuda_stream) );
    checkCudaErrors( hipMemcpyAsync(top_index_host,top_index_gpu,sizeof(int)*nBlocks_nf3,hipMemcpyDeviceToHost, _cuda_stream) );
    gettimeofday(&et4,NULL);
    _exec_logger->_cputime->t6_data_transfer_in_loop += (et4.tv_sec-et3.tv_sec + (float)((et4.tv_usec-et3.tv_usec)*1e-6));
    checkCudaErrors( hipStreamSynchronize(_cuda_stream) );

    if(num_sort!=1) {
        for(int i=0; i<nBlocks_nf3; i++) {
            if(top_index_host[i]/nf2 > nf1 || top_index_host[i] < 0){
                top_score_host[i] = -99999.99;
                //printf(" error, %d | score, %f \n", top_index_host[i]/nf2, top_score_host[i]);
            }
            const float raw = top_score_host[i];
            if( raw < _Select[num_sort-1].score) continue;
            for( int j = 0 ; j < num_sort ; j++ ) {
                if( raw > _Select[j].score ) {
                    for( int k = num_sort-1 ; k > j ; k-- ) {
                        _Select[k] = _Select[k-1];
                    }
                    const int index = top_index_host[i];
                    _Select[j].score    = raw;
                    _Select[j].index[1] = index / nf2;
                    _Select[j].index[2] = (index / _Num_fft) % _Num_fft;
                    _Select[j].index[3] = index % _Num_fft;
                    break;
                }
            }
        }

    } else { // num_sort = 1, select only 1 score per 1 ligand angle
        for(int i=0; i<nBlocks_nf3; i++) {
            if(top_index_host[i]/nf2 > nf1 || top_index_host[i] < 0){
                top_score_host[i] = -99999.99;
                //printf(" error, %d | score, %f \n", top_index_host[i]/nf2, top_score_host[i]);
            }
            if(temp_top_score < top_score_host[i]) {
                temp_top_score = top_score_host[i];
                temp_top_index = top_index_host[i];
            }
        }

        //printf("  m:%f\n\n",temp_top_score);
        //printf("%g (%d) [%d %d %d]\n", temp_top_score, _p, temp_top_index/(n*n),(temp_top_index/n)%n, temp_top_index%n );
        //printf("<%d> %g (%d/%d) %d\n", nBlocks,temp_top_score, temp_top_index, nf3, temp_top_index/nf2);

        _Select[0].score    = temp_top_score;
        _Select[0].index[1] = temp_top_index / nf2;
        _Select[0].index[2] = (temp_top_index / nf1) % nf1;
        _Select[0].index[3] = temp_top_index % nf1;
        /* / DEBUG
        printf("TEST,  %d\n", _Select[0].index[1]);
        if ( _Select[0].index[1] > nf1 ){
            printf(" error, %d\n", _Select[0].index[1]);
            }*/

    }

    //*** score_sort ***********************************************************

    for( int i = 0 ; i < num_sort ; i++ ) {
        _Select[i].index[0] = _Current_rot_angle_num;
        _Top[_Current_rot_angle_num*num_sort+i] = _Select[i];
    }

    //size_t devmem_use, devmem_free, devmem_total;
    //hipMemGetInfo(&devmem_free, &devmem_total);
    //devmem_use = devmem_total - devmem_free;
    //printf(" [GPU (%d) memory] Use : %10u (%4.1f%%), Free : %10u (%4.1f%%), Total : %10u\n",myid2,devmem_use,(float)(100*devmem_use/devmem_total), devmem_free, (float)(100*devmem_free/devmem_total), devmem_total);


    checkCudaErrors( hipStreamSynchronize(_cuda_stream) );
    gettimeofday(&et2,NULL);
    _exec_logger->_cputime->t3_5_fftprocess_score_sort += (et2.tv_sec-et1.tv_sec + (float)((et2.tv_usec-et1.tv_usec)*1e-6));

    return;
}


//============================================================================//
void FFTProcessTable::ligand_voxelization_on_gpu(float *theta, size_t myid2)
//============================================================================//
{
    const int ng1 = _Num_fft / 2;
    const int ng3 = ng1 * ng1 * ng1;
    const int nf1 = _Num_fft;
    const int nf2 = nf1 * nf1;
    const size_t nf3 = nf2 * nf1;

    const float delta = 1.0;
    const float surface = 1.0;
    const float grid_width = _parameter->grid_width;
    const int sr_half = (2.4 + grid_width - 0.01) / grid_width;
    const int sr = 2 * sr_half + 1;

    const int na = _ligand->num_atoms();
    const int nag = na * ng1;
    const int na_sr3 = na * sr * sr * sr;

    struct timeval et1, et2;
    struct timeval et3, et4;

    const int nThreads = NUM_THREADS;
    //const int nBlocks_na = (na + (nThreads-1)) / nThreads;
    const int nBlocks_nag = (nag + (nThreads-1)) / nThreads;
    const int nBlocks_na_sr3 = (na_sr3 + (nThreads-1)) / nThreads;
    const int nBlocks_ng3 = (ng3 + (nThreads-1)) / nThreads;
    const int nBlocks_nf3 = (nf3 + (nThreads-1)) / nThreads;
    if(nBlocks_nf3 * nThreads < nf3) {
        printf(" nf3:%d, nBlocks_nf3:%d, nThreads:%d , nf3=nBlocks_nf3*nThreads\n",nf3,nBlocks_nf3,nThreads);
        fprintf(stderr, " [ERROR] too large FFT size. nf3:%d, nBlocks_nf3:%d\n", nf3, nBlocks_nf3);
        exit(1);
    }

    //*
    //transfer ligand angle & calc xd,yd,zd,atom_coord_rotated
    gettimeofday(&et3,NULL);

    gettimeofday(&et1,NULL);
    checkCudaErrors( hipMemcpyAsync(ligand_rotation_angle_gpu, theta, sizeof(float)*3, hipMemcpyHostToDevice, _cuda_stream) );
    gettimeofday(&et2,NULL);
    _exec_logger->_cputime->t3_1_ligand_voxelization += (et2.tv_sec-et1.tv_sec + (float)((et2.tv_usec-et1.tv_usec)*1e-6));

    //lig_rotation<<<nBlocks_na, nThreads, 0, _cuda_stream>>>(na, ligand_rotation_angle_gpu,atom_coord_orig_gpu, mole_center_coord_gpu, atom_coord_rotated_gpu);
    //checkCudaErrors( hipStreamSynchronize(_cuda_stream) );
    //lig_calc_dis_atomgrid<<<nBlocks_nag, nThreads, 0, _cuda_stream>>>(na, ng1, xd_gpu, yd_gpu, zd_gpu, grid_coord_gpu, atom_coord_rotated_gpu);
    //checkCudaErrors( hipStreamSynchronize(_cuda_stream) );
    ligvoxgpu_copy_htod<<<nBlocks_nag, nThreads, 0, _cuda_stream>>>
        (na, ligand_rotation_angle_gpu, ng1, atom_coord_orig_gpu, mole_center_coord_gpu, atom_coord_rotated_gpu, xd_gpu, yd_gpu, zd_gpu, grid_coord_gpu);
    checkCudaErrors( hipStreamSynchronize(_cuda_stream) );
    gettimeofday(&et4,NULL);
    _exec_logger->_cputime->t3_1_1_ligvoxgpu_copy_htod += (et4.tv_sec-et3.tv_sec + (float)((et4.tv_usec-et3.tv_usec)*1e-6));

    //grid[] initialize
    gettimeofday(&et3,NULL);
    lig_vox_init<<<nBlocks_nf3, nThreads, 0, _cuda_stream>>>(ng3,nf3,grid_r_gpu,grid_i_gpu,CUFFTin_gpu);
    //lig_vox_init_fft<<<nBlocks_nf3, nThreads, 0, _cuda_stream>>>(nf3,CUFFTin_gpu);
    checkCudaErrors( hipStreamSynchronize(_cuda_stream) );
    gettimeofday(&et4,NULL);
    _exec_logger->_cputime->t3_1_2_ligvoxgpu_kernel_init += (et4.tv_sec-et3.tv_sec + (float)((et4.tv_usec-et3.tv_usec)*1e-6));

    //atom fill(core)
    gettimeofday(&et3,NULL);
    lig_vox_fill<<<nBlocks_na_sr3, nThreads, 0, _cuda_stream>>>
    (ng1,na,delta,radius_core2_gpu,xd_gpu,yd_gpu,zd_gpu,grid_coord_gpu,atom_coord_rotated_gpu,grid_r_gpu, grid_width);
    checkCudaErrors( hipStreamSynchronize(_cuda_stream) );
    gettimeofday(&et4,NULL);
    _exec_logger->_cputime->t3_1_3_ligvoxgpu_kernel_fill_core += (et4.tv_sec-et3.tv_sec + (float)((et4.tv_usec-et3.tv_usec)*1e-6));

    //surface cutting
    gettimeofday(&et3,NULL);
    lig_vox_surface_cut_CtoT<<<nBlocks_ng3, nThreads, 0, _cuda_stream>>>(ng1,delta,grid_r_gpu);
    checkCudaErrors( hipStreamSynchronize(_cuda_stream) );
    gettimeofday(&et4,NULL);
    _exec_logger->_cputime->t3_1_4_ligvoxgpu_kernel_cut_surf += (et4.tv_sec-et3.tv_sec + (float)((et4.tv_usec-et3.tv_usec)*1e-6));

    //atom fill(surf)
    gettimeofday(&et3,NULL);
    lig_vox_fill<<<nBlocks_na_sr3, nThreads, 0, _cuda_stream>>>
    (ng1,na,surface,radius_surf2_gpu,xd_gpu,yd_gpu,zd_gpu,grid_coord_gpu,atom_coord_rotated_gpu,grid_r_gpu, grid_width);
    checkCudaErrors( hipStreamSynchronize(_cuda_stream) );
    gettimeofday(&et4,NULL);
    _exec_logger->_cputime->t3_1_5_ligvoxgpu_kernel_fill_surf += (et4.tv_sec-et3.tv_sec + (float)((et4.tv_usec-et3.tv_usec)*1e-6));

    //electro
    gettimeofday(&et3,NULL);

    if(_parameter->lig_elec_serial_flag == 0) {
        lig_vox_elec<<<nBlocks_ng3, nThreads, 0, _cuda_stream>>>(ng1, na, grid_width, _Charge_gpu, atom_coord_rotated_gpu, grid_i_gpu);
    } else {
        lig_vox_elec_serial<<<nBlocks_ng3, nThreads, 0, _cuda_stream>>>(ng1, na, grid_width, _Charge_gpu, atom_coord_rotated_gpu, grid_i_gpu);
    }

    checkCudaErrors( hipStreamSynchronize(_cuda_stream) );
    gettimeofday(&et4,NULL);
    _exec_logger->_cputime->t3_1_6_ligvoxgpu_kernel_elec += (et4.tv_sec-et3.tv_sec + (float)((et4.tv_usec-et3.tv_usec)*1e-6));

    //set Voxel grid[ng3] into center of FFT grid[nf3]
    gettimeofday(&et3,NULL);
    ligand_voxel_set<<<nBlocks_ng3, nThreads, 0, _cuda_stream>>>(ng1,CUFFTin_gpu,grid_r_gpu,grid_i_gpu);
    checkCudaErrors( hipStreamSynchronize(_cuda_stream) );
    gettimeofday(&et4,NULL);
    _exec_logger->_cputime->t3_1_7_ligvoxgpu_kernel_set_array += (et4.tv_sec-et3.tv_sec + (float)((et4.tv_usec-et3.tv_usec)*1e-6));

}


//============================================================================//
void FFTProcessTable::ligand_data_transfer_gpu(float *grid_coord)
//============================================================================//
{
    const int ng1 = _Num_fft / 2;
    const int na = _ligand->num_atoms();
    const int num_gpu = _parallel->num_gpu();
    const int nproc2 = _parallel->nproc2();
    const float   rcore2 = 1.5;           // ZDOCK parameter
    const float   rsurf2 = 1.0;           // ZDOCK parameter
    struct timeval et1, et2;

    float radius_core2[na];
    float radius_surf2[na];

    for(int i = 0; i < na; i++) {
        radius_core2[i] = _ligand->_Radius[i] * _ligand->_Radius[i] * rcore2;
        radius_surf2[i] = _ligand->_Radius[i] * _ligand->_Radius[i] * rsurf2;
    }

    gettimeofday(&et1,NULL);
    int myid2 = omp_get_thread_num();
    hipSetDevice(myid2 % num_gpu);
    checkCudaErrors( hipMemcpyAsync(radius_core2_gpu, radius_core2, sizeof(float)*na, hipMemcpyHostToDevice, _cuda_stream) );
    checkCudaErrors( hipMemcpyAsync(radius_surf2_gpu, radius_surf2, sizeof(float)*na, hipMemcpyHostToDevice, _cuda_stream) );
    checkCudaErrors( hipMemcpyAsync(_Charge_gpu, _ligand->_Charge, sizeof(float)*na, hipMemcpyHostToDevice, _cuda_stream) );
    checkCudaErrors( hipMemcpyAsync(grid_coord_gpu, grid_coord, sizeof(float)*ng1, hipMemcpyHostToDevice, _cuda_stream) );
    checkCudaErrors( hipMemcpyAsync(atom_coord_orig_gpu, _ligand->_Coordinate, sizeof(float)*na*3, hipMemcpyHostToDevice, _cuda_stream) );
    checkCudaErrors( hipMemcpyAsync(mole_center_coord_gpu, _ligand->_Center, sizeof(float)*3, hipMemcpyHostToDevice, _cuda_stream) );

    gettimeofday(&et2,NULL);
    _exec_logger->_cputime->t6_data_transfer_lig += (et2.tv_sec-et1.tv_sec + (float)((et2.tv_usec-et1.tv_usec)*1e-6));

    return;
}

#endif /* CUFFT */



//============================================================================//
void FFTProcessTable::fft_memory_free()
//============================================================================//
{
    const size_t nproc2    = _parallel->nproc2();
    const int num_gpu = _parallel->num_gpu();
    const size_t nf3 = _Num_fft * _Num_fft * _Num_fft;

#ifndef CUFFT
    fftwf_destroy_plan(plan_fftw_forward);
    fftwf_destroy_plan(plan_fftw_inverse);

    _exec_logger->record_free(sizeof(float)*nf3*2);

#else

    //const int num_sort = _parameter->_Num_sort;
    const int nThreads = NUM_THREADS;
    const int nBlocks_nf3 = (nf3 + (nThreads-1)) / nThreads;

    int myid2 = omp_get_thread_num();
    hipSetDevice(myid2 % num_gpu);

    hipfftDestroy(cufft_plan);

    checkCudaErrors( hipStreamDestroy(_cuda_stream));

    checkCudaErrors( hipFree(CUFFTin_gpu));
    checkCudaErrors( hipFree(CUFFTout_gpu));
    checkCudaErrors( hipFree(_FFT_rec_r_gpu));
    checkCudaErrors( hipFree(_FFT_rec_i_gpu));


    checkCudaErrors( hipFree(grid_r_gpu));
    checkCudaErrors( hipFree(grid_i_gpu));
    checkCudaErrors( hipFree(grid_coord_gpu));

    checkCudaErrors( hipFree(radius_core2_gpu));
    checkCudaErrors( hipFree(radius_surf2_gpu));
    checkCudaErrors( hipFree(_Charge_gpu));

    checkCudaErrors( hipFree(xd_gpu));
    checkCudaErrors( hipFree(yd_gpu));

    checkCudaErrors( hipFree(zd_gpu));

    checkCudaErrors( hipFree(atom_coord_rotated_gpu));
    checkCudaErrors( hipFree(atom_coord_orig_gpu));
    checkCudaErrors( hipFree(mole_center_coord_gpu));
    checkCudaErrors( hipFree(ligand_rotation_angle_gpu));

    checkCudaErrors( hipFree(top_score_gpu));
    checkCudaErrors( hipFree(top_index_gpu));

    delete [] top_score_host;
    delete [] top_index_host;


    _exec_logger->record_free( sizeof(float)*nBlocks_nf3 + sizeof(int)*nBlocks_nf3 );

#endif

    return;
}
