#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2019 Tokyo Institute of Technology
 */

//============================================================================//
//
//  Software Name : MEGADOCK
//
//  cuda_kernel.cu
//
//  Contact address : Tokyo Institute of Technology, AKIYAMA Lab.
//
//============================================================================//

#include <stdio.h>

#define   FMAX(a,b)  ( ((a)>(b) ) ? (a) : (b) )
#define   FMIN(a,b)  ( ((a)>(b) ) ? (b) : (a) )

__global__ void lig_vox_fill(int ng1
                             ,int na
                             ,float delta
                             ,float *radius2
                             ,float *xd
                             ,float *yd
                             ,float *zd
                             ,float *grid_coord
                             ,float *atom_coord_rotated
                             ,float *grid_r
    						 ,float grid_width)
{
    int index  = blockIdx.x * blockDim.x + threadIdx.x;
    //const int ng2 = ng1 * ng1;
    const int search_range = (2.4 + grid_width - 0.01) / grid_width;
    const int sr = 2 * search_range + 1;
    const int na_sr3 = na * sr * sr * sr;
    //const int ng3 = ng2 * ng1;

    if(index < na_sr3) {
        const int k2 = index % sr - search_range;
        index /= sr;
        const int j2 = index % sr - search_range;
        index /= sr;
        const int i2 = index % sr - search_range;
        const int id = index / sr;
        const int lc = ng1 * id;
        const int id3 = id * 3;
        const int i = atom_coord_rotated[id3  ] / grid_width + ng1 / 2 + i2;
        const int j = atom_coord_rotated[id3+1] / grid_width + ng1 / 2 + j2;
        const int k = atom_coord_rotated[id3+2] / grid_width + ng1 / 2 + k2;

        if (0 <= i && i < ng1 && 0 <= j && j < ng1 && 0 <= k && k < ng1 && xd[lc + i] + yd[lc + j] + zd[lc + k] < radius2[id])
            grid_r[ng1 * (ng1 * i + j) + k] = delta;
    }
    //*/
}


__global__ void lig_rotation(int na, float *theta, float *atom_coord_orig, float *mole_center_coord, float *atom_coord_rotated)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;

    const float r11 = cos(theta[0])*cos(theta[2])  -  sin(theta[0])*cos(theta[1])*sin(theta[2]);
    const float r21 = sin(theta[0])*cos(theta[2])  +  cos(theta[0])*cos(theta[1])*sin(theta[2]);
    const float r31 = sin(theta[1])*sin(theta[2]);
    const float r12 = -cos(theta[0])*sin(theta[2])  -  sin(theta[0])*cos(theta[1])*cos(theta[2]);
    const float r22 = -sin(theta[0])*sin(theta[2])  +  cos(theta[0])*cos(theta[1])*cos(theta[2]);
    const float r32 = sin(theta[1])*cos(theta[2]);
    const float r13 = sin(theta[0])*sin(theta[1]);
    const float r23 = -cos(theta[0])*sin(theta[1]);
    const float r33 = cos(theta[1]);

    if(id < na) {
        const int id3 = id * 3;
        float     x, y, z;

        x = atom_coord_orig[id3  ] - mole_center_coord[0];
        y = atom_coord_orig[id3+1] - mole_center_coord[1];
        z = atom_coord_orig[id3+2] - mole_center_coord[2];
        atom_coord_rotated[id3  ] = r11 * x + r12 * y + r13 * z;
        atom_coord_rotated[id3+1] = r21 * x + r22 * y + r23 * z;
        atom_coord_rotated[id3+2] = r31 * x + r32 * y + r33 * z;
    }
}


__global__ void ligvoxgpu_copy_htod(const int na, const float *const theta, const int ng1, const float *const atom_coord_orig, const float *const mole_center_coord, float *atom_coord_rotated, float *xd, float *yd, float *zd, const float *const grid_coord)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    const int nag = na * ng1;
    if(id < nag) {
        const float r11 = cos(theta[0])*cos(theta[2])  -  sin(theta[0])*cos(theta[1])*sin(theta[2]);
        const float r21 = sin(theta[0])*cos(theta[2])  +  cos(theta[0])*cos(theta[1])*sin(theta[2]);
        const float r31 = sin(theta[1])*sin(theta[2]);
        const float r12 = -cos(theta[0])*sin(theta[2])  -  sin(theta[0])*cos(theta[1])*cos(theta[2]);
        const float r22 = -sin(theta[0])*sin(theta[2])  +  cos(theta[0])*cos(theta[1])*cos(theta[2]);
        const float r32 = sin(theta[1])*cos(theta[2]);
        const float r13 = sin(theta[0])*sin(theta[1]);
        const float r23 = -cos(theta[0])*sin(theta[1]);
        const float r33 = cos(theta[1]);

        const int cur_atom = (id / ng1);
        const int cur_atom3 = cur_atom * 3;
        const int cur_grid = id % ng1;

        const float x = atom_coord_orig[cur_atom3  ] - mole_center_coord[0];
        const float y = atom_coord_orig[cur_atom3+1] - mole_center_coord[1];
        const float z = atom_coord_orig[cur_atom3+2] - mole_center_coord[2];
        const float acr_x = r11 * x + r12 * y + r13 * z;
        const float acr_y = r21 * x + r22 * y + r23 * z;
        const float acr_z = r31 * x + r32 * y + r33 * z;
        if (cur_grid == 0) {
            atom_coord_rotated[cur_atom3  ] = acr_x;
            atom_coord_rotated[cur_atom3+1] = acr_y;
            atom_coord_rotated[cur_atom3+2] = acr_z;
        }

        xd[id] = acr_x - grid_coord[cur_grid];
        yd[id] = acr_y - grid_coord[cur_grid];
        zd[id] = acr_z - grid_coord[cur_grid];
        xd[id] *= xd[id];
        yd[id] *= yd[id];
        zd[id] *= zd[id];
    }
}

__global__ void lig_calc_dis_atomgrid(int na, int ng1, float *xd, float *yd, float *zd, float *grid_coord, float *atom_coord_rotated)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    const int nag = na * ng1;
    if(id < nag) {
        const int cur_atom = (id / ng1);
        const int cur_atom3 = cur_atom * 3;
        const int cur_grid = id % ng1;
        xd[id] = atom_coord_rotated[cur_atom3  ] - grid_coord[cur_grid];
        yd[id] = atom_coord_rotated[cur_atom3+1] - grid_coord[cur_grid];
        zd[id] = atom_coord_rotated[cur_atom3+2] - grid_coord[cur_grid];
        xd[id] *= xd[id];
        yd[id] *= yd[id];
        zd[id] *= zd[id];
    }
}

__global__ void lig_vox_init_grid(int ng3,float *grid_r,float *grid_i)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < ng3) { //initialize
        grid_r[id]=0.0;
        grid_i[id]=0.0;
    }
}

__global__ void lig_vox_init_fft(int nf3,hipfftComplex *lig_in)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < nf3) { //initialize
        lig_in[id] =  make_hipComplex( 0.0, 0.0);
        //lig_in[id].x=0.0;
        //lig_in[id].y=0.0;
    }
}

__global__ void lig_vox_init(int ng3,int nf3,float *grid_r,float *grid_i,hipfftComplex *lig_in)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < ng3) { //initialize
        grid_r[id]=0.0;
        grid_i[id]=0.0;
        lig_in[id] =  make_hipComplex( 0.0, 0.0);
    } else if (id < nf3) {
        lig_in[id] =  make_hipComplex( 0.0, 0.0);
    }
}

__global__ void ligand_voxel_set(int ng1
                                 ,hipfftComplex *lig_in
                                 ,float *grid_r
                                 ,float *grid_i)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    const int ng2 = ng1 * ng1;
    const int ng3 = ng2 * ng1;
    const int nf1 = ng1 * 2;
    const int nf2 = nf1 * nf1;
    const int ng1_half = ng1 / 2;
    const float surface = 1.0;        // grid-assignment score (protein surface)

    //if(id == 0) for(int i=0;i<ng3;i++) if(grid_i[i]!=0.0) printf(" [%03d,%03d,%03d] :  %6.3f\n",i/ng2,i/ng1%ng1,i%ng1,grid_i[i]);
    //if(id == 0) for(int i=0;i<ng3;i++) printf(" [%03d,%03d,%03d] :  %6.3f\n",i/ng2,(i/ng1)%ng1,i%ng1,grid_i[i]);

    if(id < ng3) {
        const int i = id / ng2;
        const int j = (id / ng1) % ng1;
        const int k = id % ng1;
        const int idoff = (i + ng1_half) * nf2 + (j + ng1_half) * nf1 + (k + ng1_half);

        //*
        if(grid_r[id]==surface) {// this condition judges whether surface(1.0) or temporary score(-8888.0)
            lig_in[idoff] =  make_hipComplex( grid_r[id], grid_i[id]);
        } else {
            lig_in[idoff] =  make_hipComplex( 0.0, grid_i[id]);
        }
        //*
    }
}


__global__ void lig_vox_surface_cut_CtoT(int ng1, float delta, float *grid_r)
{
    // Core score to Temporary score
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    const float swollen_surface = -8888.0;
    const int ng2 = ng1 * ng1;
    const int ng3 = ng2 * ng1;
    if(id < ng3) {
        const int i = id / ng2;
        const int j = (id / ng1) % ng1;
        const int k = id % ng1;
        if(i==0||i==ng1-1||j==0||j==ng1-1||k==0||k==ng1-1) { // skip border
        } else {
            if(grid_r[id]==delta) {
                if(grid_r[id-1]==0 ||
                        grid_r[id+1]==0 ||
                        grid_r[id-ng1]==0 ||
                        grid_r[id+ng1]==0 ||
                        grid_r[id-ng2]==0 ||
                        grid_r[id+ng2]==0) {
                    grid_r[id]=swollen_surface; 
                }
            }
        }
    }
}

__global__ void lig_vox_elec(int ng1,int na,float grid_width,float *_Charge,float *atom_coord_rotated,float *grid_i)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    const int ng2 = ng1 * ng1;
    const float pad = (ng1 * grid_width / 2);

    //*
    if(id < na) {
        //if(!fabs(_Charge[id]) < 0.0001) continue;
        const int id3 = id * 3;
        //const int   i = (atom_coord_rotated[id3  ] + pad) / grid_width;
        //const int   j = (atom_coord_rotated[id3+1] + pad) / grid_width;
        //const int   k = (atom_coord_rotated[id3+2] + pad) / grid_width;
        const int   i = FMAX(0, FMIN((atom_coord_rotated[id3  ] + pad) / grid_width, ng1 - 1));
        const int   j = FMAX(0, FMIN((atom_coord_rotated[id3+1] + pad) / grid_width, ng1 - 1));
        const int   k = FMAX(0, FMIN((atom_coord_rotated[id3+2] + pad) / grid_width, ng1 - 1));

        //grid_i[i*ng2+j*ng1+k] += _Charge[id];
        //printf(" %08d-1 :  %.2f, %.2f\n",i*ng2+j*ng1+k,grid_i[i*ng2+j*ng1+k],_Charge[id]);
        //if (0 <= i && i < ng1 && 0 <= j && j < ng1 && 0 <= k && k < ng1)
        atomicAdd(&grid_i[i*ng2+j*ng1+k],_Charge[id]);

        //printf(" %08d-2 :  %.2f, %.2f\n",i*ng2+j*ng1+k,grid_i[i*ng2+j*ng1+k],_Charge[id]);
    }
    //*/
}

__global__ void lig_vox_elec_serial(int ng1,int na,float grid_width,float *_Charge,float *atom_coord_rotated,float *grid_i)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    const int ng2 = ng1 * ng1;
    const float pad = (ng1 * grid_width / 2);

    if(id==0) {
        for( int l = 0 ; l < na ; l++ ) {
            const int l3 = l*3;
            const int i =(atom_coord_rotated[l3  ] + pad) / grid_width;
            const int j =(atom_coord_rotated[l3+1] + pad) / grid_width;
            const int k =(atom_coord_rotated[l3+2] + pad) / grid_width;
            //printf(" [%5d] [x:%12.8f,y:%12.8f,z:%12.8f] [pad:%6.3f], [%3d,%3d,%3d] \n",l,atom_coord_rotated[l3  ],atom_coord_rotated[l3+1],atom_coord_rotated[l3+2],pad,i,j,k);
            //printf(" [%5d] [x:%8.0f,y:%8.0f,z:%8.0f] [pad:%6.3f], [%3d,%3d,%3d] \n",l,atom_coord_rotated[l3  ],atom_coord_rotated[l3+1],atom_coord_rotated[l3+2],pad,i,j,k);

            //if(grid_i[i*ng2+j*ng1+k]!=0)printf(" Pos : %d, current : %f, new : %f\n",i*ng2+j*ng1+k, grid_i[i*ng2+j*ng1+k], _Charge[l]);

            grid_i[i*ng2+j*ng1+k] += _Charge[l];
        }
    }
}


__device__ void lig_vox_surface_cut_TtoO(int ng3, float delta, float *grid_r)
{
    // Temporary score to Open space score
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    const float swollen_surface = -8888.0; // temporary score for swollen ligand surface
    if(id < ng3) {
        if(grid_r[id]==swollen_surface) { 
            grid_r[id]=0.0;
        }
    }
}

__global__ void convolution_gpu(int nf3, float *rec_r, float *rec_i, hipfftComplex *lig_out, hipfftComplex *lig_in)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;

    if(id<nf3) {
        const float lig_r = lig_out[id].x;
        const float lig_i = lig_out[id].y;

        lig_in[id] =  make_hipComplex( rec_r[id]*lig_r + rec_i[id]*lig_i, rec_r[id]*lig_i - rec_i[id]*lig_r);
        //lig_in[id].x = rec_r[id]*lig_r + rec_i[id]*lig_i;
        //lig_in[id].y = rec_r[id]*lig_i - rec_i[id]*lig_r;
    }
}

__global__ void max_pos_single(int nf3, hipfftComplex *out, float *score, int *pos)
{
    //blockDim.x = nThreads
    //score[nBlocks], pos[nBlocks] (nBlocks = nf3 / nThreads)
    //sdata[nThreads]
    extern __shared__ float sdata[];
    float mscore;

    const int thr_id  = threadIdx.x;
    const int nThreads = blockDim.x;
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;

    if(id < nf3) {
        mscore = sdata[thr_id] = hipCrealf(out[id])/nf3;
        __syncthreads();    //all threads set sdata[thr_id]

        //reduction
        for(int offset = nThreads / 2; offset > 0; offset /= 2) {
            if (thr_id < offset) {
                sdata[thr_id] = FMAX(sdata[thr_id],  sdata[thr_id +  offset]);
            }
            __syncthreads();
        }

        if (mscore == sdata[0]) {//mscore specify position of max score
            score[blockIdx.x] = sdata[0];
            pos[blockIdx.x] = id;
            //printf("   BLOCK ID:%d, sdata[0]=%f, pos=%d\n",blockIdx.x,sdata[0],id);
        }
    }
}

__global__ void max_pos_multi_set(int nf3, hipfftComplex *out, float *temp_score, int *temp_index)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < nf3) {
        temp_score[id] = out[id].x;
        temp_index[id] = id;
    }
}

//, std::vector<hipfftComplex> *temp_result , thrust::vector<hipfftComplex> *temp_result
//thrust::device_ptr<hipfftComplex> *temp_result hipfftComplex *temp_result,thrust::device_ptr<hipfftComplex> temp_result
__global__ void max_pos_multi(int nf3, hipfftComplex *out, float *score, int *pos,const int num_sort,const int offset)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < offset) {
        if (out[id].x < out[id+offset].x) {
            out[id].x = out[id+offset].x;
            out[id].y = out[id+offset].y;
        }
        /*
        if(id==0) {
            for(int i=0; i<num_sort*2; i++) printf(" id=%d, %f %f\n",i,out[i].x/nf3,out[i].y);
            printf("\n");
        }
        //*/
    }
    //*/
}



/*
__global__ void max_pos_multi(int nf3, hipfftComplex *out, float *score, int *pos,const int num_sort, float *temp_score, int *temp_index)
{
    //blockDim.x = nThreads,
    //score[nBlocks], pos[nBlocks] (nBlocks = nf3 / nThreads)
    //sdata[nThreads]
    extern __shared__ float sdata[];
    float mscore;
    int offset;

    const int thr_id  = threadIdx.x;
    const int nThreads = blockDim.x;
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;

    //*
    if(id < nf3) {
        temp_score[id]=hipCrealf(out[id])/nf3;
        temp_index[id]=id;


        /*
        __syncthreads();    //all threads set sdata[thr_id]

        //reduction
        for(offset = nThreads / 2; offset > num_sort; ) {
            offset /= 2;
            if (thr_id < offset) {
                sdata[thr_id] = FMAX(sdata[thr_id],  sdata[thr_id +  offset]);
            }
            //if(id<1)printf(" id=%d, t=%d, off=%d\n",id,num_sort,offset);
            __syncthreads();
        }
        //if(id<1)printf(" [last] id=%d, t=%d, off=%d\n",id,num_sort,offset);

        //thrust::sort(sdata,sdata+10);

        if(id < num_sort) {
            if (mscore == sdata[id]) {//mscore specify position of max score (float equality comparison... amari yokunai)
                score[blockIdx.x] = sdata[0];
                pos[blockIdx.x] = id;
                //printf("   BLOCK ID:%d, sdata[0]=%f, pos=%d\n",blockIdx.x,sdata[0],i);
            }
        }
        //*
        if(temp_score[id] >3000) printf(" id=%d, %f %d\n",id,temp_score[id],temp_index[id]);
    }
    //*
}
//*/


