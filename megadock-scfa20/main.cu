/*
 * Copyright (C) 2014 Tokyo Institute of Technology
 *
 *
 * This file is part of MEGADOCK.
 * MEGADOCK is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MEGADOCK is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MEGADOCK.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

//============================================================================//
//
//  Software Name : MEGADOCK
//
//  Class Name : (main)
//
//  Contact address : Tokyo Institute of Technology, AKIYAMA Lab.
//
//============================================================================//

#include <string.h>
#include "cpu_time.h"
#include "exec_logger.h"
#include "control_pdb.h"
#include "control_table.h"

#ifdef CUFFT
#include <hip/hip_runtime_api.h>
#define VERSION "4.1.0 for GPU & "
#else
#define VERSION "4.1.0 for CPU & "
#endif

#ifdef MPI_DP
#define VTEXT "multiple nodes"
#else
#define VTEXT "single node"
#endif

#define LASTUPDATED "26 March, 2019"

struct DockingPair {
    string rec_file, lig_file, out_file;
    DockingPair(string rec_file, string lig_file, string out_file) : rec_file(rec_file), lig_file(lig_file), out_file(out_file) {}
};

//============================================================================//
void get_pair(string line, string &rec_file, string &lig_file, string &out_file)
//============================================================================//
{
    int first_tab_index = line.find_first_of('\t');
    if (first_tab_index == string::npos) {
        cerr << "[Error] Ligand is not specified." << endl;
        exit(1);
    }
    rec_file = line.substr(0, first_tab_index);

    int second_tab_index = line.find_last_of('\t');
    if (first_tab_index == second_tab_index) {
        lig_file = line.substr(first_tab_index + 1, line.size() - 1 - first_tab_index);
        out_file = "";
    } else {
        lig_file = line.substr(first_tab_index + 1, second_tab_index - 1 - first_tab_index);
        out_file = line.substr(second_tab_index + 1, line.size() - 1 - second_tab_index);
    }
}

//============================================================================//
void initialize(int argc, char *argv[], int &nproc2, int &device_count_gpu)
//============================================================================//
{
    cout << " MEGADOCK ver. "<< VERSION << VTEXT <<  endl;
    cout << "      megadock@bi.c.titech.ac.jp   lastupdated: " << LASTUPDATED << endl;
    cout << endl;

#ifdef _OPENMP
    #pragma omp parallel
    {
        nproc2 = omp_get_num_threads();
        if(omp_get_thread_num() == 0) {
            cout << "# Using OpenMP parallelization: " << nproc2 << " threads." << endl;
        }
    }
    //printf("#OpenMP version %d\n", _OPENMP);
#else
    nproc2 = 1;
#endif //#ifdef _OPENMP

#ifdef CUFFT
    int nogpu_flag = 0;
    for (int num = 0; num < (argc-1); ++num) {
        if(!strncmp(argv[num], "-G", 2)) {
            if(argv[num+1] != NULL) {
                if(atoi(argv[num+1]) == 0) {
                    nogpu_flag = 1;
                }
            }
        }
    }

    if(nogpu_flag != 1) {
        checkCudaErrors( hipGetDeviceCount(&device_count_gpu) );
        if (device_count_gpu == 0) {
            fprintf(stderr, "GPU Error: no devices supporting CUDA.\n");
            exit(-1);
        }

        hipDeviceProp_t deviceProp;
        checkCudaErrors( hipGetDeviceProperties(&deviceProp, 0));
        if (deviceProp.major < 1) {
            fprintf(stderr, "GPU Error: device does not support CUDA.\n");
            exit(-1);
        }

        hipSetDeviceFlags(hipDeviceMapHost);
        fprintf(stdout, "# Using CUDA device %d: %s\n", 0, deviceProp.name);
        hipSetDevice(0);
        //fprintf(stdout, "# Init CUDA device OK.\n");

        int cufft_version;
        hipfftGetVersion(&cufft_version);
        printf("# CUFFT version : %d\n", cufft_version);
    }

    printf("# Number of available [threads / GPUs] : [%d / %d]\n",nproc2,device_count_gpu);
#endif
}

//============================================================================//
void main_pdb(int argc, char *argv[])
//============================================================================//
{
    Parallel  *_parallel;
    CPUTime   *_cputime;
    ControlPDB   *_control;

    struct timeval et1, et2;
    struct timeval et3, et4;
    int nproc2 = 0;
    int device_count_gpu = 0;

    gettimeofday(&et1,NULL);
    gettimeofday(&et3,NULL);

    initialize(argc, argv, nproc2, device_count_gpu);

    _cputime = new CPUTime();
    _cputime->initialize();

    _parallel = new Parallel(nproc2);
    _parallel->num_gpu(device_count_gpu); 

    gettimeofday(&et4,NULL);
    _cputime->t1_initialize += (et4.tv_sec-et3.tv_sec + (float)((et4.tv_usec-et3.tv_usec)*1e-6));

    _control = new ControlPDB(_cputime,_parallel);
    _control->initialize(argc,argv);
    _control->execute();

    delete _control;
    delete _parallel;

    _cputime->output();

    delete _cputime;

    gettimeofday(&et2,NULL);

    const float elapsed_time = (et2.tv_sec-et1.tv_sec + (float)((et2.tv_usec-et1.tv_usec)*1e-6));
    printf("\n");
    printf("Elapsed time                  = %8.2f sec.\n",elapsed_time);
}

//============================================================================//
void main_table(int argc, char *argv[])
//============================================================================//
{
    struct timeval et3, et4;
    int nproc2 = 0;
    int device_count_gpu = 0;

    gettimeofday(&et3,NULL);

    initialize(argc, argv, nproc2, device_count_gpu);

    struct timeval et1[nproc2], et2[nproc2];

    Parallel  *_parallels[nproc2];
    ExecLogger   *_exec_loggers[nproc2];
    ControlTable   *_controls[nproc2];
    ParameterTable *_parameters[nproc2];

    for (int i = 0; i < nproc2; i++) {
        _parallels[i] = new Parallel(nproc2);
        _parallels[i]->num_gpu(device_count_gpu);
        _exec_loggers[i] = new ExecLogger();

        // ParameterTable
        _parameters[i] = new ParameterTable(_parallels[i]);
        if (i == 0) {
            _parameters[i]->initialize(argc,argv);
        } else {
            _parameters[i]->initialize(_parameters[0]);
        }
        _exec_loggers[i]->record_malloc(_parameters[i]->allocate_size()); //Rotation angles[], Atom radius, charge, ACE[]

        _controls[i] = new ControlTable(_exec_loggers[i],_parallels[i],_parameters[i]);
        _controls[i]->initialize(i == 0);
    }


    ifstream input_stream(_controls[0]->input_file());
    if (!input_stream.is_open()) {
        cerr << "Unable to open input file." << endl;
        exit(1);
    }
    string line;
    vector<DockingPair> pairs;
    while (getline(input_stream, line)) {
        string rec_file, lig_file, out_file;
        get_pair(line, rec_file, lig_file, out_file);
        pairs.push_back(DockingPair(rec_file, lig_file, out_file));
    }

#pragma omp parallel for schedule(dynamic, 1)
    for (int i = 0; i < pairs.size(); i++) {
        int myid2 = omp_get_thread_num();
        DockingPair _docking_pair = pairs[i];
        gettimeofday(&et1[myid2],NULL);
        _exec_loggers[myid2]->initialize();
#pragma omp critical (prepare)
        {
            _controls[myid2]->prepare(_docking_pair.rec_file, _docking_pair.lig_file, _docking_pair.out_file);
        }
        _controls[myid2]->execute();

        gettimeofday(&et2[myid2],NULL);

        const float elapsed_time = (et2[myid2].tv_sec-et1[myid2].tv_sec + (float)((et2[myid2].tv_usec-et1[myid2].tv_usec)*1e-6));
        printf("\n");

#pragma omp critical (output)
        {
            printf("# ========================================\n");
            _exec_loggers[myid2]->output(myid2);
            printf("Elapsed time                  = %8.2f sec.\n"
                   "# ========================================\n"
                   ,elapsed_time);
        }
    }

#pragma omp parallel for
    for (int i = 0; i < nproc2; i++) {
        delete _exec_loggers[i];
        delete _controls[i];
        delete _parallels[i];
        delete _parameters[i];
    }

    gettimeofday(&et4,NULL);

    const float total_time = (et4.tv_sec-et3.tv_sec + (float)((et4.tv_usec-et3.tv_usec)*1e-6));
    printf("\n");
    printf("Total time                    = %8.2f sec.\n",total_time);
}

//============================================================================//
#ifdef MPI_DP
int application(int argc,char *argv[])
#else
int main(int argc, char *argv[])
#endif
//============================================================================//
{
    bool table_input_flag = false, pdb_input_flag = false;
    for (int num = 0; num < argc; ++num) {
        if (!(strncmp(argv[num], "-R", 2) && strncmp(argv[num], "-L", 2) && strncmp(argv[num], "-o", 2))) {
            pdb_input_flag = true;
        } else if (!strncmp(argv[num], "-I", 2)) {
            table_input_flag = true;
        } else if (!strncmp(argv[num], "-h", 2)) {
            usage();
        }
    }
    if (pdb_input_flag) {
        if (table_input_flag) {
            fprintf(stderr, "[ERROR] A pair of PDB files and a docking pair list file cannot be specified simultaneously.\n");
            usage();
        } else {
            main_pdb(argc, argv);
        }
    } else {
        if (table_input_flag) {
            main_table(argc, argv);
        } else {
            fprintf(stderr, "[ERROR] A pair of PDB files or a docking pair list file has to be specified.\n");
            usage();
        }
    }
    return 0;
}
