/*
 * Copyright (C) 2014 Tokyo Institute of Technology
 *
 *
 * This file is part of MEGADOCK.
 * MEGADOCK is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MEGADOCK is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MEGADOCK.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

//============================================================================//
//
//  Software Name : MEGADOCK
//
//  Class Name : Application
//
//  Contact address : Tokyo Institute of Technology, AKIYAMA Lab.
//
//============================================================================//

#include "application.h"

//============================================================================//
void Application::initialize()
//============================================================================//
{
#ifdef CUFFT
    checkCudaErrors( hipGetDeviceCount(&device_count_gpu) );
    if (device_count_gpu == 0) {
        fprintf(stderr, "GPU Error: no devices supporting CUDA.\n");
        exit(-1);
    }

    hipDeviceProp_t deviceProp;
    checkCudaErrors( hipGetDeviceProperties(&deviceProp, 0));
    if (deviceProp.major < 1) {
        fprintf(stderr, "GPU Error: device does not support CUDA.\n");
        exit(-1);
    }

    hipSetDeviceFlags(hipDeviceMapHost);
    fprintf(stdout, "# Using CUDA device %d: %s\n", 0, deviceProp.name);
    hipSetDevice(0);
    //fprintf(stdout, "# Init CUDA device OK.\n");

    int cufft_version;
    hipfftGetVersion(&cufft_version);
    printf("# CUFFT version : %d\n", cufft_version);

    printf("# Number of available [threads / GPUs] : [%d / %d]\n",nproc2,device_count_gpu);
#endif

    _parallels = new Parallel*[nproc2];
    _exec_loggers = new ExecLogger*[nproc2];
    _controls = new ControlTable*[nproc2];
    _parameters = new ParameterTable*[nproc2];

    for (int i = 0; i < nproc2; i++) {
        _parallels[i] = new Parallel(nproc2);
        _parallels[i]->num_gpu(device_count_gpu);
        _exec_loggers[i] = new ExecLogger();

        // ParameterTable
        _parameters[i] = new ParameterTable(_parallels[i]);
        if (i == 0) {
            _parameters[i]->initialize();
        } else {
            _parameters[i]->initialize(_parameters[0]);
        }
        _exec_loggers[i]->record_malloc(_parameters[i]->allocate_size()); //Rotation angles[], Atom radius, charge, ACE[]

        _controls[i] = new ControlTable(_exec_loggers[i],_parallels[i],_parameters[i]);
        _controls[i]->initialize(i == 0);
    }
}

//============================================================================//
int Application::application(int argc, char *argv[], int myid2)
//============================================================================//
{
    struct timeval et1, et2;
    gettimeofday(&et1,NULL);
    _exec_loggers[myid2]->initialize();
#pragma omp critical (prepare)
    {
        _parameters[myid2]->process_args(argc, argv);
        _controls[myid2]->prepare();
    }
    _controls[myid2]->execute();

    gettimeofday(&et2,NULL);

    const float elapsed_time = (et2.tv_sec-et1.tv_sec + (float)((et2.tv_usec-et1.tv_usec)*1e-6));
    printf("\n");

#pragma omp critical (output)
    {
        printf("# ========================================\n");
        _exec_loggers[myid2]->output(myid2);
        printf("Elapsed time                  = %8.2f sec.\n"
               "# ========================================\n"
               ,elapsed_time);
    }
    return 0;
}
